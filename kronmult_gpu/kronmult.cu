#include "hip/hip_runtime.h"
#include <>
#include "kronmult.cuh"

/*
 * computes number^power for integers
 * does not care about performances
 * does not use std::pow as it does an implicit float conversion that could lead to rounding errors for large numbers
 */
__host__ int pow_int(const int number, const int power)
{
    if(power == 0) return 1;
    return number * pow_int(number, power-1);
}

/*
 * Computes output += kron(matrix_list) * input while insuring that the addition to output is thread-safe
 *
 * `matrix_list` is an array containing pointers to `matrix_number` square matrices of size `matrix_size` by `matrix_size` and stride `matrix_stride`
 * `input` is a `size_input` (`matrix_size`^`matrix_number`) elements vector
 * `output` is a `size_input` elements vector, to which the output of the multiplication will be added
 * `workspace` is a `size_input` elements vector, to be used as workspace
 * `transpose_workspace` is a vector of size `matrix_size`*`matrix_size` to store transposed matrices temporarily
 *
 * WARNINGS:
 * - `input`, `workspace` and `transpose_workspace` will be used as temporary workspaces and thus modified
 * - the matrices are assumed to be stored in col-major order
 * - the sizes are assumed to be correct
 */
template<typename T>
__device__ void cuda_kronmult(const int matrix_count, const int matrix_size, T const * const matrix_list[], const int matrix_stride,
                              T input[], const int size_input,
                              T output[], T workspace[], T transpose_workspace[])
{
    // how many column should `input` have for the multiplications to be legal
    const int nb_col_input = size_input / matrix_size;

    // iterates on the matrices from last to first
    for(int i = matrix_count-1; i >= 0; i--)
    {
        // transpose the matrix (with a single thead) to get a better memory coalescing
        T const * const matrix = matrix_list[i];
        if(threadIdx.x == 0) transpose(matrix, transpose_workspace, matrix_size, matrix_stride);
        __syncthreads();

        // performs the multiplication to consume the matrix
        multiply_transpose<T>(input, nb_col_input, transpose_workspace, matrix_size, workspace);
        __syncthreads();

        // swap `input` and `workspace` such that `input` contains once again the input
        // note that, while they have the same size flattened, the shape (nb_columns and nb_rows) of `input` and `workspace` are different
        // this is on purpose and equivalent to a reshape operation that is actually needed by the algorithm
        T* temp = input;
        input = workspace;
        workspace = temp;
    }

    // adds result to output in a thread-safe way
    // strided loop, each thread threadIdx.x manages the input i such that i % threadIdx.x==0
    for(int i = threadIdx.x; i < size_input; i+=blockDim.x)
    {
        atomicAdd(&output[i], input[i]);
    }
}

/*
 * each block gets a single batch element to process
 *
 * computes the current batch element
 * finds the corresponding inputs
 * and calls kronmult on them
 */
template<typename T>
__global__ void cuda_kronmult_batchelement(const int matrix_count, const int matrix_size, T const * const matrix_list_batched[], const int matrix_stride,
                                           T* input_batched[], const int size_input,
                                           T* output_batched[], T* workspace_batched[],
                                           const int nb_batch)
{
    // each block corresponds to a single batch element
    const int batchId = blockIdx.x;
    // gets the inputs for a given batch element
    T const * const * matrix_list = &matrix_list_batched[batchId*matrix_count];
    T* input = input_batched[batchId];
    T* output = output_batched[batchId];
    T* workspace = workspace_batched[batchId];

    // uses a thread to allocates the transpose workspace
    // in shared memory for improved performances
    __shared__ T* transpose_workspace;
    if(threadIdx.x == 0) transpose_workspace = new T[matrix_size*matrix_size];
    __syncthreads();

    // does the kronmult computations
    cuda_kronmult<T>(matrix_count, matrix_size, matrix_list, matrix_stride, input, size_input, output, workspace, transpose_workspace);

    // frees the tranpose workspace memory
    __syncthreads();
    if(threadIdx.x == 0) delete[] transpose_workspace;
}

/*
 * calls the cuda kernel with the proper number of blocks and threads
 * we expect the inputs to already be on the GPU
 */
template<typename T>
__host__ hipError_t cuda_kronmult_batched(const int matrix_count, const int matrix_size, T const * const matrix_list_batched[], const int matrix_stride,
                                         T* input_batched[], T* output_batched[], T* workspace_batched[], const int nb_batch)
{
    // numbers of elements in the input vector
    int size_input = pow_int(matrix_size, matrix_count);

    // each block will take care of a single batch element
    // the threads within a block will loop over input_size
    int deviceId;
    hipGetDevice(&deviceId);
    int threadsPerBlock;
    hipDeviceGetAttribute(&threadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, deviceId);
    if(size_input < threadsPerBlock) threadsPerBlock = size_input;

    // parallelize over batch elements
    cuda_kronmult_batchelement<<<nb_batch, threadsPerBlock>>>(matrix_count, matrix_size, matrix_list_batched, matrix_stride,
                                                              input_batched, size_input, output_batched, workspace_batched, nb_batch);

    // waits for kernel to finish and returns the error code
    return hipDeviceSynchronize();
}

/*
 * double specialization of kronmult_batched
 */
template<>
__host__ hipError_t kronmult_batched<double>(const int matrix_count, const int matrix_size, double const * const matrix_list_batched[], const int matrix_stride,
                                            double* input_batched[], double* output_batched[], double* workspace_batched[], const int nb_batch)
{
    return cuda_kronmult_batched(matrix_count, matrix_size, matrix_list_batched, matrix_stride, input_batched, output_batched, workspace_batched, nb_batch);
}

/*
 * float specialization of kronmult_batched
 */
template<>
__host__ hipError_t kronmult_batched<float>(const int matrix_count, const int matrix_size, float const * const matrix_list_batched[], const int matrix_stride,
                                           float* input_batched[], float* output_batched[], float* workspace_batched[], const int nb_batch)
{
    return cuda_kronmult_batched(matrix_count, matrix_size, matrix_list_batched, matrix_stride, input_batched, output_batched, workspace_batched, nb_batch);
}
